
#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>


void dp0(double* a, double* b, long N, double& dp) {
	//compute the dot product of a and b using the cpu

	double sum = 0;
	#pragma omp parallel for reduction(+:sum)
	for (long i = 0; i < N; i++) {
		sum += a[i]*b[i];
	}
}

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

#define BLOCK_SIZE 1024

__global__ void reduction_kernel2(double* sum, const double* a, long N){
	//reduction kernel for summing
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

__global__ void mult_kernel(double* a, double* b, double* c, long N) {
	//cuda kernel to compute pairwise multiplication of a and b
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N) {
		c[idx] = a[idx] * b[idx];
	}
}


void dot(double* a, double* b, long N, double& dp) {
	//take in a and b vectors, apply dot product and reduction kernels. 

	//allocate a vector for the product
	double *c_d;
	hipMalloc(&c_d, N*sizeof(double));

	//call the multiplication kernel
	mult_kernel<<<N/BLOCK_SIZE+1,BLOCK_SIZE>>>(a, b, c_d, N);

	//call reduction kernel on c
	double *y_d;
  hipMalloc(&y_d, ((N+BLOCK_SIZE-1)/BLOCK_SIZE)*sizeof(double));
	double* sum_d = y_d;
  long Nb = (N+BLOCK_SIZE-1)/(BLOCK_SIZE);
  reduction_kernel2<<<Nb,BLOCK_SIZE>>>(sum_d, c_d, N);
  while (Nb > 1) {
    long N = Nb;
    Nb = (Nb+BLOCK_SIZE-1)/(BLOCK_SIZE);
    reduction_kernel2<<<Nb,BLOCK_SIZE>>>(sum_d + Nb, sum_d, N);
    sum_d += Nb;
  }


  hipMemcpyAsync(&dp, sum_d, 1*sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();





	//free memory
	hipFree(c_d);
}



















int main() {
  long N = (1UL<<10); //10 was 25
  //long N = 100;

  //initialize vector
  double *v;
  hipHostMalloc((void **) &v, N * sizeof(double), hipHostMallocDefault);
  //#pragma omp parallel for 
  for (long i = 0; i < N; i++) {
  	printf("test\n");
    v[i] = 1.0/(i+1);
  }

  //initialize matrix
  double* a;
  hipHostMalloc((void **) &a, N*N*sizeof(double), hipHostMallocDefault);
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N*N; i++) {
  	a[i] = drand48(); 
  }

  //get a reference solution for dot product
  double dp;
  double dp_ref;
  dp0(v,v,N,dp_ref);

  //get a reference solution for matrix vector product
  double* mult;
  double* mult_ref;
  hipHostMalloc((void **) &mult_ref, N * sizeof(double), hipHostMallocDefault);
  //mv0(a, v, N, mult_ref);

  //copy memory to gpu
  double *v_d, *a_d, *mult_d;
  hipMalloc(&v_d, N*sizeof(double));
  hipMalloc(&a_d, N*N*sizeof(double));
  hipMalloc(&mult_d, N*sizeof(double));

  hipMemcpyAsync(v_d, v, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(a_d, a, N*N*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  //do dot product on gpu
  dot(v_d, v_d, N, dp);

  //get error
  double errDP = fabs(dp_ref-dp);
  printf("Dot product Error: %f", errDP);


  //free memory
  hipFree(v_d);
  hipFree(a_d);
  hipHostFree(v);
  hipHostFree(a);
  hipHostFree(mult_ref); hipHostFree(mult);

  

  return 0;
}